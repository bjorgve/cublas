#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <Eigen/Dense>
#include <vector>

#define m 6
#define n 6
#define k 6
#define lda m
#define ldb k
#define ldc m

// Function to print a matrix
void print_matrix(Eigen::MatrixXcf matrix) {
    for(int i = 0; i < matrix.rows(); i++) {
        for(int j = 0; j < matrix.cols(); j++) {
            printf("(%3.1f, %3.1f) ", matrix(i, j).real(), matrix(i, j).imag());
        }
        printf("\n");
    }
}

int main(void) {
    hipblasHandle_t handle;
    Eigen::MatrixXcf a = Eigen::MatrixXcf::Random(m, k);
    Eigen::MatrixXcf b = Eigen::MatrixXcf::Random(k, n);
    Eigen::MatrixXcf c = Eigen::MatrixXcf::Zero(m, n);

    printf("Matrix A:\n");
    print_matrix(a);

    printf("\nMatrix B:\n");
    print_matrix(b);

    // Convert Eigen matrices to std::vector<hipComplex>
    std::vector<hipComplex> a_data(a.size());
    std::vector<hipComplex> b_data(b.size());
    std::vector<hipComplex> c_data(c.size());
    for (int i = 0; i < a.rows(); ++i) {
        for (int j = 0; j < a.cols(); ++j) {
            a_data[j * a.rows() + i] = make_hipComplex(a(i, j).real(), a(i, j).imag());
            b_data[j * b.rows() + i] = make_hipComplex(b(i, j).real(), b(i, j).imag());
        }
    }

    // Allocate device memory
    hipComplex* d_a;
    hipMalloc((void**)&d_a, m*k*sizeof(hipComplex));
    hipComplex* d_b;
    hipMalloc((void**)&d_b, k*n*sizeof(hipComplex));
    hipComplex* d_c;
    hipMalloc((void**)&d_c, m*n*sizeof(hipComplex));

    // Initialize the CUBLAS library
    hipblasCreate(&handle);

    // Copy matrices to the device
    hipblasSetMatrix(m, k, sizeof(hipComplex), a_data.data(), lda, d_a, lda);
    hipblasSetMatrix(k, n, sizeof(hipComplex), b_data.data(), ldb, d_b, ldb);

    hipComplex al = make_hipComplex(1.0f, 0.0f);
    hipComplex bet = make_hipComplex(0.0f, 0.0f);

    // Perform operation using cublas
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, lda, d_b, ldb, &bet, d_c, ldc);

    // Retrieve the result matrix C from the device
    hipblasGetMatrix(m, n, sizeof(hipComplex), d_c, ldc, c_data.data(), ldc);

    // Convert result back to Eigen matrix
    for (int i = 0; i < c.rows(); ++i) {
        for (int j = 0; j < c.cols(); ++j) {
            c(i, j) = std::complex<float>(c_data[j * c.rows() + i].x, c_data[j * c.rows() + i].y);
        }
    }

    printf("\nResult Matrix C:\n");
    print_matrix(c);

    // Destroy the handle
    hipblasDestroy(handle);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return EXIT_SUCCESS;
}
