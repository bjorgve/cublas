#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define m 6
#define n 6
#define k 6
#define lda m
#define ldb k
#define ldc m

// Function to print a matrix
void print_matrix(hipComplex *matrix, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            printf("(%3.1f, %3.1f) ", matrix[i + j*lda].x, matrix[i + j*lda].y);
        }
        printf("\n");
    }
}

int main(void) {
    hipblasHandle_t handle;
    hipComplex* a;
    hipComplex* b;
    hipComplex* c;
    a = (hipComplex *)malloc(m * k * sizeof(hipComplex));
    b = (hipComplex *)malloc(k * n * sizeof(hipComplex));
    c = (hipComplex *)malloc(m * n * sizeof(hipComplex));

    // Initialize the matrices here...
    for(int i = 0; i < m*k; i++){
        a[i].x = i + 1;
        a[i].y = i + 1;
    }

    for(int i = 0; i < k*n; i++){
        b[i].x = i + 1;
        b[i].y = i + 1;
    }

    printf("Matrix A:\n");
    print_matrix(a, m, k);

    printf("\nMatrix B:\n");
    print_matrix(b, k, n);

    // Allocate device memory
    hipComplex* d_a;
    hipMalloc((void**)&d_a, m*k*sizeof(hipComplex));
    hipComplex* d_b;
    hipMalloc((void**)&d_b, k*n*sizeof(hipComplex));
    hipComplex* d_c;
    hipMalloc((void**)&d_c, m*n*sizeof(hipComplex));

    // Initialize the CUBLAS library
    hipblasCreate(&handle);

    // Copy matrices to the device
    hipblasSetMatrix(m, k, sizeof(hipComplex), a, lda, d_a, lda);
    hipblasSetMatrix(k, n, sizeof(hipComplex), b, ldb, d_b, ldb);

    hipComplex al = make_hipComplex(1.0f, 0.0f);
    hipComplex bet = make_hipComplex(0.0f, 0.0f);

    // Perform operation using cublas
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &al, d_a, lda, d_b, ldb, &bet, d_c, ldc);

    // Retrieve the result matrix C from the device
    hipblasGetMatrix(m, n, sizeof(hipComplex), d_c, ldc, c, ldc);

    printf("\nResult Matrix C:\n");
    print_matrix(c, m, n);

    // Destroy the handle
    hipblasDestroy(handle);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return EXIT_SUCCESS;
}
