#include "matrix_mul_gpu.h"

// Function Definition
void multiply_matrices_gpu(EigenMat& matA, EigenMat& matB, EigenMat& matC){
    // Check if matrix dimensions match
    assert(matA.cols() == matB.rows());

    // Extract the matrix dimensions
    int m = matA.rows(); // Number of rows in op(A) and C.
    int n = matB.cols(); // Number of columns in op(B) and C.
    int k = matA.cols(); // Number of columns in op(A) and rows in op(B).

    // Check if the output matrix has the correct dimensions
    assert(matC.rows() == m);
    assert(matC.cols() == n);
    assert(k == matB.rows());


    // Set the leading dimensions
    // Rows for column-major ordering
    int lda = matA.rows();
    int ldb = matB.rows();
    int ldc = matC.rows();

    // Create a cuBlas handle.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Create device pointers.
    hipDoubleComplex* d_A;
    hipDoubleComplex* d_B;
    hipDoubleComplex* d_C;


    auto matA_size = matA.rows()*matA.cols()*sizeof(hipDoubleComplex);
    hipMalloc((void**)&d_A, matA_size);
    hipMemcpy(d_A, matA.data(), matA_size, hipMemcpyHostToDevice);

    auto matB_size = matB.rows()*matB.cols()*sizeof(hipDoubleComplex);
    hipMalloc((void**)&d_B, matB_size);
    hipMemcpy(d_B, matB.data(), matB_size, hipMemcpyHostToDevice);

    auto matC_size = matC.rows()*matC.cols()*sizeof(hipDoubleComplex);
    hipMalloc((void**)&d_C, matC_size);

    // Constants for hipblasZgemm routine.
    const hipDoubleComplex alf = make_hipDoubleComplex(1,0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0,0);

    // Perform multiplication.
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alf, d_A, lda, d_B, ldb, &bet, d_C, ldc);

    // Copy the result back to the output Eigen matrix.
    hipMemcpy(matC.data(), d_C, matC_size, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy the cuBlas handle.
    hipblasDestroy(handle);
}
