#include "matrix_mul_gpu.h"

// Function Definition
void multiply_matrices_gpu(EigenMat& matA, EigenMat& matB, EigenMat& matC){
    // Check if matrix dimensions match
    assert(matA.cols() == matB.rows());

    // Extract the matrix dimensions
    int ar = matA.rows();    // number of rows in the A matrix
    int ac = matA.cols();    // number of columns in the A matrix (rows in B)
    int bc = matB.cols();    // number of columns in the B matrix

    // Create a cuBlas handle.
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Create device pointers.
    hipDoubleComplex* d_A;
    hipDoubleComplex* d_B;
    hipDoubleComplex* d_C;

    hipMalloc((void**)&d_A, ar*ac*sizeof(hipDoubleComplex));
    hipMemcpy(d_A, matA.data(), ar*ac*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_B, ac*bc*sizeof(hipDoubleComplex));
    hipMemcpy(d_B, matB.data(), ac*bc*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_C, ar*bc*sizeof(hipDoubleComplex));

    // Constants for hipblasZgemm routine.
    const hipDoubleComplex alf = make_hipDoubleComplex(1,0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0,0);

    // Perform multiplication.
    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ar, bc, ac, &alf, d_A, ar, d_B, ac, &bet, d_C, ar);

    // Copy the result back to the output Eigen matrix.
    hipMemcpy(matC.data(), d_C, ar*bc*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy the cuBlas handle.
    hipblasDestroy(handle);
}
